#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMVisualisationHelpers_CUDA.h"

using namespace ITMLib;

//device implementations
/**
 * @brief 统计可见的entry列表中，block id在指定范围内的数量
 * @param[in] visibleEntryIDs   可见entry的id列表。
 * @param[in] noVisibleEntries  可见entry数量
 * @param[in] hashTable         hash table
 * @param[out] noBlocks         指定范围内的entry数量
 * @param[in] minBlockId        指定id范围
 * @param[in] maxBlockId        定id范围
 */
__global__ void ITMLib::countVisibleBlocks_device(const int *visibleEntryIDs, int noVisibleEntries,
                                                  const ITMHashEntry *hashTable, uint *noBlocks, int minBlockId,
                                                  int maxBlockId) {
  int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (globalIdx >= noVisibleEntries)
    return;

  int entryId = visibleEntryIDs[globalIdx];
  int blockId = hashTable[entryId].ptr;
  if ((blockId >= minBlockId) && (blockId <= maxBlockId)) // 只统计在指定范围内的可见entry数量
    atomicAdd(noBlocks, 1); // TODO: 应该用归约的方式求和
}

__global__ void ITMLib::buildCompleteVisibleList_device(
    const ITMHashEntry *hashTable, /*ITMHashCacheState *cacheStates, bool useSwapping,*/
    int noTotalEntries, int *visibleEntryIDs, int *noVisibleEntries, uchar *entriesVisibleType, Matrix4f M,
    Vector4f projParams, Vector2i imgSize, float voxelSize) {
  int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (targetIdx > noTotalEntries - 1)
    return;

  __shared__ bool shouldPrefix;
  // 只支持0、1两种可见类型。没有那么多花里胡哨的
  unsigned char hashVisibleType = 0; // entriesVisibleType[targetIdx];
  const ITMHashEntry &hashEntry = hashTable[targetIdx];

  shouldPrefix = false;
  __syncthreads();

  if (hashEntry.ptr >= 0) { // 对存在的voxel block检查可见性（不将图片扩大尺寸）
    shouldPrefix = true;

    bool isVisible, isVisibleEnlarged;
    checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M, projParams, voxelSize, imgSize);

    hashVisibleType = isVisible;
  }
  // 可见的就记录一下
  if (hashVisibleType > 0)
    shouldPrefix = true;  // 只要cuda block中有一个线程要记录，整个cuda block就参与后面的前缀和计算
  __syncthreads();

  if (shouldPrefix) { // 通过前缀求和找到全局数组中的位置
    int offset =
        computePrefixSum_device<int>(hashVisibleType > 0, noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
    if (offset != -1)
      visibleEntryIDs[offset] = targetIdx;
  }
}
/**
 * @brief 将可见的voxel block投影到当前相机视角 && 分小块记录最大最小深度。用来辅助后续raycast
 * @param[in] hashEntries hash table
 * @param[in] visibleEntryIDs 所有可见的entry的id
 * @param[in] noVisibleEntries 所有可见的entry的总数
 * @param[in] pose_M 当前相机位姿。world to local
 * @param[in] intrinsics 相机内参
 * @param[in] imgSize 成像的图片大小
 * @param[in] voxelSize 真实的voxel size。单位米
 * @param[out] renderingBlocks voxel block投影到成像平面后的分块
 * @param[out] noTotalBlocks 上面分块的总数。
 * @note 这里找到的最大最小深度就是后面raycast的搜索范围。分块可以更加精细地确定深度范围，从而减少后面raycast的搜索
 */
__global__ void ITMLib::projectAndSplitBlocks_device(const ITMHashEntry *hashEntries, const int *visibleEntryIDs,
                                                     int noVisibleEntries, const Matrix4f pose_M,
                                                     const Vector4f intrinsics, const Vector2i imgSize, float voxelSize,
                                                     RenderingBlock *renderingBlocks, uint *noTotalBlocks) {
  int in_offset = threadIdx.x + blockDim.x * blockIdx.x;

  const ITMHashEntry &blockData(hashEntries[visibleEntryIDs[in_offset]]);
  //! 将单个可见的block投影到 当前视角下，并计算包围盒 && 深度范围
  Vector2i upperLeft, lowerRight;   // 包围盒的左上、右下坐标
  Vector2f zRange;
  bool validProjection = false;
  if (in_offset < noVisibleEntries) // TODO:in_offset>noVisibleEntries应该return
    if (blockData.ptr >= 0)         // >=0表示当前block有效 
      validProjection = ProjectSingleBlock(blockData.pos, pose_M, intrinsics, imgSize, voxelSize, upperLeft, 
                                           lowerRight, zRange);
  //! 将包围盒分小块，每块大小(renderingBlockSizeX,renderingBlockSizeY)=(16,16)。ceilf是向上取整。为啥要分块渲染？？？
  Vector2i requiredRenderingBlocks(ceilf((float)(lowerRight.x - upperLeft.x + 1) / renderingBlockSizeX),
  ceilf((float) (lowerRight.y - upperLeft.y + 1) / renderingBlockSizeY));

  size_t requiredNumBlocks = requiredRenderingBlocks.x * requiredRenderingBlocks.y; // 包围盒中小块数量
      // TODO: 按照renderingBlockSizeX和renderingBlockSizeY都为16，不可能有requiredNumBlocks>1
  if (!validProjection) requiredNumBlocks = 0;    // TODO：直接return 就好？还是为了一定要有下面的computePrefixSum_device？
  // 通过前缀和 来找到每个小块 在最终数组里的位置
  int out_offset = computePrefixSum_device<uint>(requiredNumBlocks, noTotalBlocks, blockDim.x, threadIdx.x);
  if (!validProjection) return;
  if ((out_offset == -1) || (out_offset + requiredNumBlocks > MAX_RENDERING_BLOCKS)) return;

  CreateRenderingBlocks(renderingBlocks, out_offset, upperLeft, lowerRight, zRange);  // 创建小块
}

__global__ void ITMLib::checkProjectAndSplitBlocks_device(const ITMHashEntry *hashEntries, int noHashEntries,
                                                          const Matrix4f pose_M, const Vector4f intrinsics,
                                                          const Vector2i imgSize, float voxelSize,
                                                          RenderingBlock *renderingBlocks, uint *noTotalBlocks) {
  int targetIdx = threadIdx.x + blockDim.x * blockIdx.x;
  if (targetIdx >= noHashEntries) return;

  const ITMHashEntry &hashEntry = hashEntries[targetIdx];

  Vector2i upperLeft, lowerRight;
  Vector2f zRange;
  bool validProjection = false;
  if (hashEntry.ptr >= 0)
    validProjection = ProjectSingleBlock(hashEntry.pos,
                                         pose_M,
                                         intrinsics,
                                         imgSize,
                                         voxelSize,
                                         upperLeft,
                                         lowerRight,
                                         zRange);

  Vector2i requiredRenderingBlocks(ceilf((float)(lowerRight.x - upperLeft.x + 1) / renderingBlockSizeX),
  ceilf((float) (lowerRight.y - upperLeft.y + 1) / renderingBlockSizeY));
  size_t requiredNumBlocks = requiredRenderingBlocks.x * requiredRenderingBlocks.y;
  if (!validProjection) requiredNumBlocks = 0;

  int out_offset = computePrefixSum_device<uint>(requiredNumBlocks, noTotalBlocks, blockDim.x, threadIdx.x);
  if (requiredNumBlocks == 0) return;
  if ((out_offset == -1) || (out_offset + requiredNumBlocks > MAX_RENDERING_BLOCKS)) return;

  CreateRenderingBlocks(renderingBlocks, out_offset, upperLeft, lowerRight, zRange);
}

__global__ void ITMLib::fillBlocks_device(uint noTotalBlocks, const RenderingBlock *renderingBlocks,
                                          Vector2i imgSize, Vector2f *minmaxData) {
  int x = threadIdx.x;
  int y = threadIdx.y;
  int block = blockIdx.x * 4 + blockIdx.y;
  if (block >= noTotalBlocks) return;

  const RenderingBlock &b(renderingBlocks[block]);
  int xpos = b.upperLeft.x + x;
  if (xpos > b.lowerRight.x) return;
  int ypos = b.upperLeft.y + y;
  if (ypos > b.lowerRight.y) return;

  Vector2f &pixel(minmaxData[xpos + ypos * imgSize.x]);
  atomicMin(&pixel.x, b.zRange.x);
  atomicMax(&pixel.y, b.zRange.y);
}
/**
 * 找到渲染图片中需要重新raycast的像素
 * @param[out] fwdProjMissingPoints 所有需要重新raycast的像素的一维坐标
 * @param[out] noMissingPoints      重新raycast的像素总数
 * @param[in] minmaximg             深度范围图。判断是否需要重新raycast的条件之一
 * @param[in] forwardProjection     旧的raycast在当前视角下的投影
 * @param[in] currentDepth          当前深度图。判断是否需要重新raycast的条件之一
 * @param[in] imgSize               图像大小
 */
__global__ void ITMLib::findMissingPoints_device(int *fwdProjMissingPoints, uint *noMissingPoints,
                                                 const Vector2f *minmaximg, Vector4f *forwardProjection,
                                                 float *currentDepth, Vector2i imgSize) {
  int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

  if (x >= imgSize.x || y >= imgSize.y)
    return;

  int locId = x + y * imgSize.x;
  Vector4f fwdPoint = forwardProjection[locId];
  float depth = currentDepth[locId];
  // 获取当前像素对应ray 在之前获取的深度范围。注意：深度范围图 比 渲染图片 缩小了 minmaximg_subsample倍
  int locId2 = (int)floor((float)x / minmaximg_subsample) + (int)floor((float)y / minmaximg_subsample) * imgSize.x;
  Vector2f minmaxval = minmaximg[locId2];

  bool hasPoint = false;

  __shared__ bool shouldPrefix; // 来判断当前cuda block是否要参与前缀和计算
  shouldPrefix = false;
  __syncthreads();
  // 需要重新raycast的像素 = 找不到voxel && (旧的raycast中没有能投影到 || 深度值有效) && 存在表面   // TODO: float不应该用==0
  if ((fwdPoint.w <= 0) && ((fwdPoint.x == 0 && fwdPoint.y == 0 && fwdPoint.z == 0) || (depth > 0)) &&
      (minmaxval.x < minmaxval.y)) {  // NOTE：.w是voxel的置信度(<=0说明没有voxel），minmaxval.x < .y表示ray找到了表面
  // if ((fwdPoint.w <= 0) && (minmaxval.x < minmaxval.y))
    shouldPrefix = true;        // 只要有一个cuda thread要参与，则整个cuda block都要参与前缀和计算
    hasPoint = true;
  }

  __syncthreads();
  // 通过前缀和计算 在数组中的全局坐标
  if (shouldPrefix) {
    int offset = computePrefixSum_device(hasPoint, noMissingPoints, blockDim.x * blockDim.y,
                                         threadIdx.x + threadIdx.y * blockDim.x);
    if (offset != -1)
      fwdProjMissingPoints[offset] = locId;
  }
}
/**
 * 获取 在当前帧能找到的 上一帧raycast结果
 * @param[out] forwardProjection  在当前帧能找到的
 * @param[in] pointsRay           上一帧的raycast结果(voxel坐标)
 * @param[in] imgSize             图像大小
 * @param[in] M                   当前帧的位姿
 * @param[in] projParams          相机内参
 * @param[in] voxelSize           voxel size
 */
__global__ void ITMLib::forwardProject_device(Vector4f *forwardProjection, const Vector4f *pointsRay, Vector2i imgSize,
                                              Matrix4f M, Vector4f projParams, float voxelSize) {
  int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

  if (x >= imgSize.x || y >= imgSize.y)
    return;

  int locId = x + y * imgSize.x;
  Vector4f pixel = pointsRay[locId];  // 第四个维度是从voxel中顺带读出来的置信度

  int locId_new = forwardProjectPixel(pixel * voxelSize, M, projParams, imgSize); // 将三维点投影，获取一维像素坐标
  if (locId_new >= 0)
    forwardProjection[locId_new] = pixel;
}
