#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "../../../ORUtils/CUDADefines.h"
#include "../../Utils/ITMCUDAUtils.h"
#include "../Shared/ITMDepthTracker_Shared.h"
#include "ITMDepthTracker_CUDA.h"

using namespace ITMLib;

struct ITMDepthTracker_CUDA::AccuCell {
  int numPoints;
  float f;
  float g[6];
  float h[6 + 5 + 4 + 3 + 2 + 1];
};

struct ITMDepthTracker_KernelParameters {
  ITMDepthTracker_CUDA::AccuCell *accu;
  float *depth;
  Matrix4f approxInvPose;
  Vector4f *pointsMap;
  Vector4f *normalsMap;
  Vector4f sceneIntrinsics;
  Vector2i sceneImageSize;
  Matrix4f scenePose;
  Vector4f viewIntrinsics;
  Vector2i viewImageSize;
  float distThresh;
};

template <bool shortIteration, bool rotationOnly>
__global__ void depthTrackerOneLevel_g_rt_device(ITMDepthTracker_KernelParameters para);

// host methods

ITMDepthTracker_CUDA::ITMDepthTracker_CUDA(Vector2i imgSize, TrackerIterationType *trackingRegime,
                                           int noHierarchyLevels, float terminationThreshold,
                                           float failureDetectorThreshold, const ITMLowLevelEngine *lowLevelEngine)
    : ITMDepthTracker(imgSize, trackingRegime, noHierarchyLevels, terminationThreshold, failureDetectorThreshold,
                      lowLevelEngine, MEMORYDEVICE_CUDA) {
  ORcudaSafeCall(hipHostMalloc((void **)&accu_host, sizeof(AccuCell)));
  ORcudaSafeCall(hipMalloc((void **)&accu_device, sizeof(AccuCell)));
}

ITMDepthTracker_CUDA::~ITMDepthTracker_CUDA(void) {
  ORcudaSafeCall(hipHostFree(accu_host));
  ORcudaSafeCall(hipFree(accu_device));
}

int ITMDepthTracker_CUDA::ComputeGandH(float &f, float *nabla, float *hessian, Matrix4f approxInvPose) {
  //! 从scene中获取所需信息
  Vector4f *pointsMap = sceneHierarchyLevel->pointsMap->GetData(MEMORYDEVICE_CUDA);   // 三维坐标点。第4个数字是w？？？
  Vector4f *normalsMap = sceneHierarchyLevel->normalsMap->GetData(MEMORYDEVICE_CUDA); // 法向量。第4个数字是w？？？
  Vector4f sceneIntrinsics = sceneHierarchyLevel->intrinsics;                         // 场景相机内参
  Vector2i sceneImageSize = sceneHierarchyLevel->pointsMap->noDims;                   // 场景图像大小
  //! 从当前帧获取所需信息
  float *depth = viewHierarchyLevel->data->GetData(MEMORYDEVICE_CUDA); // 获取当前帧的深度图（以一维存储）
  Vector4f viewIntrinsics = viewHierarchyLevel->intrinsics;            // 当前帧相机内参
  Vector2i viewImageSize = viewHierarchyLevel->data->noDims;           // 当前帧图像大小

  if (iterationType == TRACKER_ITERATION_NONE)
    return 0;
  //! 初始化
  bool shortIteration =
      (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);

  int noPara = shortIteration ? 3 : 6;  // CPU版本中的noParaSQ放在下面的核函数
  
  //! 计算每个像素的 误差、Hessian矩阵 和 Nabla算子
  dim3 blockSize(16, 16);
  dim3 gridSize((int)ceil((float)viewImageSize.x / (float)blockSize.x),
                (int)ceil((float)viewImageSize.y / (float)blockSize.y));
  ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));

  struct ITMDepthTracker_KernelParameters args;
  args.accu = accu_device;
  args.depth = depth;
  args.approxInvPose = approxInvPose;
  args.pointsMap = pointsMap;
  args.normalsMap = normalsMap;
  args.sceneIntrinsics = sceneIntrinsics;
  args.sceneImageSize = sceneImageSize;
  args.scenePose = scenePose;
  args.viewIntrinsics = viewIntrinsics;
  args.viewImageSize = viewImageSize;
  args.distThresh = distThresh[levelId];

  switch (iterationType) {
  case TRACKER_ITERATION_ROTATION:    // 只跟踪 旋转
    depthTrackerOneLevel_g_rt_device<true, true><<<gridSize, blockSize>>>(args);
    ORcudaKernelCheck;
    break;
  case TRACKER_ITERATION_TRANSLATION: // 只跟踪 平移
    depthTrackerOneLevel_g_rt_device<true, false><<<gridSize, blockSize>>>(args);
    ORcudaKernelCheck;
    break;
  case TRACKER_ITERATION_BOTH:        // 跟踪 旋转+平移
    depthTrackerOneLevel_g_rt_device<false, false><<<gridSize, blockSize>>>(args);
    ORcudaKernelCheck;
    break;
  default:
    break;
  }

  ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));

  //! 获得最终的Hessian矩阵、Nabla算子 和 误差
  for (int r = 0, counter = 0; r < noPara; r++)   // H的下三角
    for (int c = 0; c <= r; c++, counter++)
      hessian[r + c * 6] = accu_host->h[counter]; // 改成hessian[r+c*6] = hessian[c+r*6] = sumHessian[counter]？？？
  for (int r = 0; r < noPara; ++r)                // H的上三角
    for (int c = r + 1; c < noPara; c++)
      hessian[r + c * 6] = hessian[c + r * 6];

  memcpy(nabla, accu_host->g, noPara * sizeof(float));
  f = (accu_host->numPoints > 100) ? accu_host->f / accu_host->numPoints : 1e5f;  //最终的误差函数

  return accu_host->numPoints;
}

// device functions
/**
 * 计算单个像素的Hessian矩阵、Nabla算子 和 误差
 * @tparam shortIteration     只求解 旋转 or 平移
 * @tparam rotationOnly       
 * @param[out] accu           计算结果，包含Hessian矩阵、Nabla算子、误差
 * @param[in] depth
 * @param[in] approxInvPose   初始位姿，=上一帧的位姿？？？
 * @param[in] pointsMap       场景投影出来的三维点？？？
 * @param[in] normalsMap      投影点对应的法向量
 * @param[in] sceneIntrinsics 场景的
 * @param[in] sceneImageSize
 * @param[in] scenePose
 * @param[in] viewIntrinsics  当前帧的
 * @param[in] viewImageSize
 * @param[in] distThresh      距离阈值，用来剔除误差大的点
 */
template <bool shortIteration, bool rotationOnly>
__device__ void depthTrackerOneLevel_g_rt_device_main(ITMDepthTracker_CUDA::AccuCell *accu, float *depth,
                                                      Matrix4f approxInvPose, Vector4f *pointsMap, Vector4f *normalsMap,
                                                      Vector4f sceneIntrinsics, Vector2i sceneImageSize,
                                                      Matrix4f scenePose, Vector4f viewIntrinsics,
                                                      Vector2i viewImageSize, float distThresh) {
  int x = threadIdx.x + blockIdx.x * blockDim.x, 
      y = threadIdx.y + blockIdx.y * blockDim.y;
  int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

  __shared__ float dim_shared1[256];
  __shared__ float dim_shared2[256];
  __shared__ float dim_shared3[256];
  __shared__ bool should_prefix;
  should_prefix = false;
  __syncthreads();

  //! 转最小二乘问题Ax=b
  const int noPara = shortIteration ? 3 : 6;
  const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
  float A[noPara];
  float b;
  bool isValidPoint = false;

  if (x < viewImageSize.x && y < viewImageSize.y) {
    isValidPoint = computePerPointGH_Depth_Ab<shortIteration, rotationOnly>(
        A, b, x, y, depth[x + y * viewImageSize.x], viewImageSize, viewIntrinsics, sceneImageSize, sceneIntrinsics,
        approxInvPose, scenePose, pointsMap, normalsMap, distThresh);
    if (isValidPoint)
      should_prefix = true;
  }
  //! 无效的像素退出
  if (!isValidPoint) {
    for (int i = 0; i < noPara; i++)
      A[i] = 0.0f;
    b = 0.0f;
  }
  __syncthreads();
  if (!should_prefix) return;

  //! 有效像素数量求和的规约
  { // reduction for noValidPoints
    dim_shared1[locId_local] = isValidPoint;
    __syncthreads();

    if (locId_local < 128)    // 128是因为block size = (16, 16)。 TODO：可以用for循环替换？
      dim_shared1[locId_local] += dim_shared1[locId_local + 128];
    __syncthreads();
    if (locId_local < 64)
      dim_shared1[locId_local] += dim_shared1[locId_local + 64];
    __syncthreads();

    if (locId_local < 32)
      warpReduce(dim_shared1, locId_local);

    if (locId_local == 0)
      atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
  }

  //! 误差求和的规约
  { // reduction for energy function value
    dim_shared1[locId_local] = b * b;
    __syncthreads();

    if (locId_local < 128)
      dim_shared1[locId_local] += dim_shared1[locId_local + 128];
    __syncthreads();
    if (locId_local < 64)
      dim_shared1[locId_local] += dim_shared1[locId_local + 64];
    __syncthreads();

    if (locId_local < 32)
      warpReduce(dim_shared1, locId_local);

    if (locId_local == 0)
      atomicAdd(&(accu->f), dim_shared1[locId_local]);
  }

  __syncthreads();

  // reduction for nabla
  //! nabla算子（高斯牛顿中的g）求和的规约
  for (unsigned char paraId = 0; paraId < noPara; paraId += 3) {
    dim_shared1[locId_local] = b * A[paraId + 0];
    dim_shared2[locId_local] = b * A[paraId + 1];
    dim_shared3[locId_local] = b * A[paraId + 2];
    __syncthreads();

    if (locId_local < 128) {
      dim_shared1[locId_local] += dim_shared1[locId_local + 128];
      dim_shared2[locId_local] += dim_shared2[locId_local + 128];
      dim_shared3[locId_local] += dim_shared3[locId_local + 128];
    }
    __syncthreads();
    if (locId_local < 64) {
      dim_shared1[locId_local] += dim_shared1[locId_local + 64];
      dim_shared2[locId_local] += dim_shared2[locId_local + 64];
      dim_shared3[locId_local] += dim_shared3[locId_local + 64];
    }
    __syncthreads();

    if (locId_local < 32) {
      warpReduce(dim_shared1, locId_local);
      warpReduce(dim_shared2, locId_local);
      warpReduce(dim_shared3, locId_local);
    }
    __syncthreads();

    if (locId_local == 0) {
      atomicAdd(&(accu->g[paraId + 0]), dim_shared1[0]);
      atomicAdd(&(accu->g[paraId + 1]), dim_shared2[0]);
      atomicAdd(&(accu->g[paraId + 2]), dim_shared3[0]);
    }
  }

  __syncthreads();

  float localHessian[noParaSQ];
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
  for (unsigned char r = 0, counter = 0; r < noPara; r++) {
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
    for (int c = 0; c <= r; c++, counter++)
      localHessian[counter] = A[r] * A[c];
  }

  // reduction for hessian
  //! Hessian矩阵求和的规约  // TODO：可以优化？？？
  for (unsigned char paraId = 0; paraId < noParaSQ; paraId += 3) {
    dim_shared1[locId_local] = localHessian[paraId + 0];
    dim_shared2[locId_local] = localHessian[paraId + 1];
    dim_shared3[locId_local] = localHessian[paraId + 2];
    __syncthreads();

    if (locId_local < 128) {
      dim_shared1[locId_local] += dim_shared1[locId_local + 128];
      dim_shared2[locId_local] += dim_shared2[locId_local + 128];
      dim_shared3[locId_local] += dim_shared3[locId_local + 128];
    }
    __syncthreads();
    if (locId_local < 64) {
      dim_shared1[locId_local] += dim_shared1[locId_local + 64];
      dim_shared2[locId_local] += dim_shared2[locId_local + 64];
      dim_shared3[locId_local] += dim_shared3[locId_local + 64];
    }
    __syncthreads();

    if (locId_local < 32) {
      warpReduce(dim_shared1, locId_local);
      warpReduce(dim_shared2, locId_local);
      warpReduce(dim_shared3, locId_local);
    }
    __syncthreads();

    if (locId_local == 0) {
      atomicAdd(&(accu->h[paraId + 0]), dim_shared1[0]);
      atomicAdd(&(accu->h[paraId + 1]), dim_shared2[0]);
      atomicAdd(&(accu->h[paraId + 2]), dim_shared3[0]);
    }
  }
}

/*只是做了个等价调用，就不注释了*/
template <bool shortIteration, bool rotationOnly>
__global__ void depthTrackerOneLevel_g_rt_device(ITMDepthTracker_KernelParameters para) {
  depthTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly>(
      para.accu, para.depth, para.approxInvPose, para.pointsMap, para.normalsMap, para.sceneIntrinsics,
      para.sceneImageSize, para.scenePose, para.viewIntrinsics, para.viewImageSize, para.distThresh);
}
